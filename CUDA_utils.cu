#include "include/Macros.hh"


/* ================================================
 * Wrapper routine around hipMalloc() for integers
 * ================================================ */
int *allocate_int_device(const int    &rank,
                         const size_t &size) {
    int *device_ptr;
    CHECK_ERROR_CUDA(rank, hipMalloc((void**) &device_ptr, size));
    return device_ptr;
}


/* ===================================
 * Wrapper routine around hipMemcpy()
 * =================================== */
void copy_device(const int            &rank,
                       void           *dest,
                 const void           *src,
                 const size_t         &size,
                 const hipMemcpyKind &kind) {
    CHECK_ERROR_CUDA(rank, hipMemcpy(dest, src, size, kind));
    return;
}


/* =================================
 * Wrapper routine around hipFree()
 * ================================= */
void free_device(const int  &rank,
                       void *device_ptr) {
    CHECK_ERROR_CUDA(rank, hipFree(device_ptr));
    return;
}
