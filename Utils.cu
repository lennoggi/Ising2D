/* ===========================================================================
 * This file contains wrappers around CUDA routines, which cannot be compiled
 * with regular C++ compilers. We could, in principle, compile everything with
 * nvcc, but not all of the modern C++ features are supported and the MPI
 * library must be linked manually.
 * NOTE: the nvc++ compiler understands CUDA routines and kernels, but for
 *       maximum portability all the CUDA-related routines are here, and this
 *       file must be compiled using nvcc.
 * =========================================================================== */
#include <hiprand/hiprand_kernel.h>
#include "include/Declare_functions.hh"
#include "include/Macros.hh"

using namespace std;


/* ===================================
 * Wrapper routine around hipMalloc()
 * =================================== */
template <typename T>
T *allocate_device(const int    &rank,
                   const size_t &num_elements) {
    T *device_ptr;
    CHECK_ERROR_CUDA(rank, hipMalloc(reinterpret_cast<void**>(&device_ptr), num_elements*sizeof(T)));
    return device_ptr;
}

template int*
allocate_device<int>(const int    &rank,
                     const size_t &size);

template hiprandStatePhilox4_32_10_t*
allocate_device<hiprandStatePhilox4_32_10_t>(const int    &rank,
                                            const size_t &size);


/* ===================================
 * Wrapper routine around hipMemcpy()
 * =================================== */
template <typename T>
void copy_device(const int    &rank,
                       T      *dest,
                       T      *src,
                 const size_t &num_elements,
                 const hipMemcpyKind &copy_kind) {
    CHECK_ERROR_CUDA(rank, hipMemcpy(reinterpret_cast<void*>(dest), reinterpret_cast<void*>(src),
                                      num_elements*sizeof(T), copy_kind));
    return;
}

template void
copy_device<int>(const int    &rank,
                       int    *dest,
                       int    *src,
                 const size_t &num_elements,
                 const hipMemcpyKind &copy_kind);



/* =====================================
 * Wrapper routine around hipMemcpy2D()
 * ===================================== */
template <typename T>
void copy_device_2D(const int    &rank,
                          T      *dest,
                    const size_t &dest_stride,  // Number of elements between successive elements in the destination memory chunk. Usually set to 1 if contiguous or to n_columns if not.
                          T      *src,
                    const size_t &src_stride,   // Number of elements between successive elements in the source memory chunk. Usually set to 1 if contiguous or to n_columns if not.
                    const size_t &width,        // Number of elements per row to copy
                    const size_t &height,       // Number of rows to copy
                    const hipMemcpyKind &copy_kind) {
    CHECK_ERROR_CUDA(rank, hipMemcpy2D(reinterpret_cast<void*>(dest), dest_stride*sizeof(T),
                                        reinterpret_cast<void*>(src),   src_stride*sizeof(T),
                                        width*sizeof(T), height, copy_kind));
    return;
}

template void
copy_device_2D<int>(const int    &rank,
                          int    *dest,
                    const size_t &dest_stride,
                          int    *src,
                    const size_t &src_stride,
                    const size_t &width,
                    const size_t &height,
                    const hipMemcpyKind &copy_kind);



/* =================================
 * Wrapper routine around hipFree()
 * ================================= */
void free_device(const int  &rank,
                       void *device_ptr) {
    CHECK_ERROR_CUDA(rank, hipFree(device_ptr));
    return;
}
