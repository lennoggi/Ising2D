#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include "include/Declare_functions.hh"
#include "include/Macros.hh"


/* =======================================================================
 * Actual kernel initializing the RNG on the device (i.e., wrapper routine
 * around hiprand_init())
 * ======================================================================= */
template <typename T> __global__
void init_rng_device_kernel(T *rng_states_device,
                            const size_t seed,
                            const size_t nx,
                            const size_t ny) {
    const auto i  = blockIdx.x*blockDim.x + threadIdx.x;
    const auto j  = blockIdx.y*blockDim.y + threadIdx.y;

    /* Safety step: make sure the threads don't access memory they shouldn't
     * NOTE: this should never happen if there aren't more threads than points
     *   in the process-local lattice along any given dimension, which we
     *   enforce in include/Check_parameters.hh                                 */
    if (i >= nx or j >= ny) {
        return;
    }

    const auto ij = i*ny + j;

    // 0 is the offset in the sequence of pseudo/quasi-random numbers generated
    hiprand_init(seed, ij, 0, &rng_states_device[ij]);  // hiprand_init() returns void

    return;
}

/* NOTE: explicit instantiation of init_rng_device_kernel() must appear BEFORE
 *   the kernel is used (used in init_rng_device() below)                       */
// TODO: add more RNG types
template __global__ void
init_rng_device_kernel<hiprandStatePhilox4_32_10_t>(hiprandStatePhilox4_32_10_t *rng_states_device,
                                                   const size_t seed,
                                                   const size_t nx,
                                                   const size_t ny);



/* ===========================================================
 * Wrapper routine around init_rng_device_kernel() (see below)
 * =========================================================== */
template <typename T>
void init_rng_device(const int &rank,
                     T *rng_states_device,
                     const size_t &seed,
                     const size_t &nx,
                     const size_t &ny,
                     const size_t &block_size_x,
                     const size_t &block_size_y) {
    // Shape of the CUDA thread block
    dim3 block(block_size_x, block_size_y);

    // Shape of the CUDA block grid
    dim3 grid((nx + block.x - 1)/block.x,   // block.x == block_size_x
              (ny + block.y - 1)/block.y);  // block.y == block_size_y

    init_rng_device_kernel<T><<<grid, block>>>(rng_states_device, seed, nx, ny);
    CHECK_ERROR_CUDA(rank, hipDeviceSynchronize());

    return;
}

// TODO: add more RNG types
template void
init_rng_device<hiprandStatePhilox4_32_10_t>(const int &rank,
                                            hiprandStatePhilox4_32_10_t *rng_states_device,
                                            const size_t &seed,
                                            const size_t &nx,
                                            const size_t &ny,
                                            const size_t &block_size_x,
                                            const size_t &block_size_y);
